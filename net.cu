#include "hip/hip_runtime.h"
#include "net.h"

#define RED 0
#define GRN 1
#define BLU 2

typedef struct GPUClassify {
  size_t *topo;
  double *activs;
  double *wgts; 
} GPUClassify_T;

struct Classify {
  size_t size;
  size_t maxNrn;
  GPUClassify_T *dev;
};

size_t findMax(size_t *, size_t);

__global__ void CNN_convolve(Convlvd_T *conv, Features_T *kern, double *img) {
  size_t x = flat2d(blockIdx.x, threadIdx.x, blockDim.x);
  size_t y = flat2d(blockIdx.y, threadIdx.y, blockDim.y);
  size_t z = flat2d(blockIdx.z, threadIdx.z, blockDim.z);

  size_t imgWid = conv->wid + kern->wid - 1;
  size_t kernPxls = kern->hgt * kern->wid;
  double chnlTotal[NUM_CHNL];
  
  if (x < kern->num) {
    if (y < conv->hgt) {
      if (z < conv->wid) {
        chnlTotal[RED] = 0.00f;
        chnlTotal[GRN] = 0.00f;
        chnlTotal[BLU] = 0.00f;
        for (size_t i = y; i - y < kern->hgt; i++) {
          for (size_t j = z; j - z < kern->wid; j++) {
            for (size_t k = 0; k < NUM_CHNL; k++) {
              chnlTotal[k] += img[flat3d(i, j, k, imgWid, NUM_CHNL)] * 
                        kern->imgs[flat4d(x, (i - y), (j - z), k, kern->hgt, kern->wid, NUM_CHNL)];
            }
          }
        }

        for (uint8_t i = 0; i < NUM_CHNL; i++) {
          conv->imgs[flat4d(x, y, z, i, conv->hgt, conv->wid, NUM_CHNL)] = chnlTotal[i] / kernPxls;
        }
      }
    }
  }
}

__global__ void cuda_initConvlvd(Convlvd_T *conv, Features_T *kern, size_t imgHgt, size_t imgWid) {
  conv->num = kern->num;
  conv->hgt = imgHgt - kern->hgt + 1;
  conv->wid = imgWid - kern->wid + 1;

  hipMalloc((void **)&conv->imgs, conv->num * conv->hgt * conv->wid * NUM_CHNL * sizeof(double));
  size_t totalPxls = kern->num * kern->hgt * kern->wid * NUM_CHNL;
  for (size_t i = 0; i < totalPxls; i++) {
    conv->imgs[i] = 0.5f;
  }
}

Convlvd_T *CNN_initConvlvd(Features_T *kern, Data_T *data) {
  Convlvd_T *conv;
  hipMalloc((void **)&conv, sizeof(Convlvd_T));
  cuda_initConvlvd<<<1, 1>>>(conv, kern, data->hgt, data->wid);
  hipDeviceSynchronize();

  return conv;
}

__global__ void cuda_freeConvlvd(Convlvd_T *conv) {
  hipFree(conv->imgs);
}

void CNN_freeConvlvd(Convlvd_T *conv) {
  cuda_freeConvlvd<<<1, 1>>>(conv);
  hipDeviceSynchronize();

  hipFree(conv);
}

__global__ void cuda_testData(double *imgs, size_t *lbls, size_t idx, size_t hgt, size_t wid) {
  printf("Red Channel:\n");
  for (size_t i = 0; i < hgt; i++) {
    for (size_t j = 0; j < wid; j++) {
      printf("%0.2f ", imgs[flat4d(idx, i, j, 0, hgt, wid, NUM_CHNL)]);
    }
    printf("\n");
  }

  printf("Green Channel:\n");
  for (size_t i = 0; i < hgt; i++) {
    for (size_t j = 0; j < wid; j++) {
      printf("%0.2f ", imgs[flat4d(idx, i, j, 1, hgt, wid, NUM_CHNL)]);
    }
    printf("\n");
  }

  printf("Blue Channel:\n");
  for (size_t i = 0; i < hgt; i++) {
    for (size_t j = 0; j < wid; j++) {
      printf("%0.2f ", imgs[flat4d(idx, i, j, 2, hgt, wid, NUM_CHNL)]);
    }
    printf("\n");
  }
}

void CNN_testData(Data_T *data, size_t idx) {
  cuda_testData<<<1, 1>>>(data->imgs, data->lbls, idx, data->hgt, data->wid);
  hipDeviceSynchronize();
}

void CNN_freeData(Data_T *data) {
  hipFree(data->lbls);
  hipFree(data->imgs);
  free(data);
}

__global__ void cuda_initFtrs(Features_T *kern, size_t num, size_t hgt, size_t wid) {
  kern->num = num;
  kern->hgt = hgt;
  kern->wid = wid;

  size_t numImg = num * hgt * wid * NUM_CHNL;
  hipMalloc((void **)&kern->imgs, numImg * sizeof(double));
  for (size_t i = 0; i < numImg; i++) {
    kern->imgs[i] = 0.5f;
  }
}

Features_T *CNN_initFtrs(size_t numFeat, size_t hgt, size_t wid) {
  Features_T *kern;
  hipMalloc((void **)&kern, sizeof(Features_T));
  cuda_initFtrs<<<1, 1>>>(kern, numFeat, hgt, wid);
  hipDeviceSynchronize();

  return kern;
}

__global__ void cuda_freeFtrs(Features_T *kern) {
  hipFree(kern->imgs);
}

void CNN_freeFtrs(Features_T *kern) {
  cuda_freeFtrs<<<1, 1>>>(kern);
  hipDeviceSynchronize();

  hipFree(kern);
}

__global__ void cuda_initClsfier(GPUClassify_T *net, size_t *topo, size_t netSize) {
  hipMalloc((void **)&net->topo, netSize * sizeof(size_t));
  /* deep copy- host memory can only be freed in host */
  for (size_t i = 0; i < netSize; i++) {
    net->topo[i] = topo[i];
  }

  size_t totalNrn = 0;
  for (size_t i = 0; i < netSize; i++) {
    totalNrn += topo[i];
  }
  hipMalloc((void **)&net->activs, totalNrn * sizeof(double));
  for (size_t i = 0; i < totalNrn; i++) {
    net->activs[i] = 0.5f;
  }

  size_t totalWgt = 0;
  for (size_t i = 0; i < netSize - 1; i++) {
    totalWgt += topo[i] * topo[i + 1];
  }
  hipMalloc((void **)&net->wgts, totalWgt * sizeof(double));
  for (size_t i = 0; i < totalWgt; i++) {
    net->wgts[i] = 0.5f;
  }
}

Classify_T *CNN_initClsfier(size_t *topology, size_t netSize) {
  Classify_T *net = (Classify_T *)malloc(sizeof(Classify_T));
  net->maxNrn = findMax(topology, netSize);
  net->size = netSize;

  hipMalloc((void **)&net->dev, sizeof(GPUClassify_T));

  /* init topology */
  size_t *topo_d;
  hipMalloc((void **)&topo_d, netSize * sizeof(size_t));
  hipMemcpy(topo_d, topology, netSize * sizeof(size_t), hipMemcpyHostToDevice);

  cuda_initClsfier<<<1,1>>>(net->dev, topo_d, netSize);
  hipDeviceSynchronize();
  hipFree(topo_d);

  return net;
}

__global__ void cuda_testClsfier(GPUClassify_T *net, size_t size) {
	for (size_t i = 0; i < size; i++) {
		printf("  layer %lu:\n", i);
		printf("    numNrn %lu:\n", net->topo[i]);
		for (size_t j = 0; j < net->topo[i]; j++) {
			printf("    neuron %lu (activ: %.2f) weights:\n      ", j, net->activs[flat2d(i, j, size)]);
      if (i != size - 1) {
				for (size_t k = 0; k < net->topo[i + 1]; k++) {
					printf("[%.2f] ", net->wgts[flat3d(i, j, k, size, net->topo[i])]);
				}
      }
			printf("\n");
		}
	}
}

void CNN_testClsfier(Classify_T *net) {
  cuda_testClsfier<<<1,1>>>(net->dev, net->size);
  hipDeviceSynchronize();
}

__global__ void cuda_freeClsfier(GPUClassify_T *net) {
  hipFree(net->topo);
  hipFree(net->activs);
  hipFree(net->wgts);
}

void CNN_freeClsfier(Classify_T *net) {
  cuda_freeClsfier<<<1,1>>>(net->dev);
  hipDeviceSynchronize();
  hipFree(net->dev);
  free(net);
}

size_t findMax(size_t *arr, size_t len) {
  size_t max = 0;
  for (size_t i = 0; i < len; i++) {
    if (arr[i] > max) {
      max = arr[i];
    }
  }

  return max;
}
