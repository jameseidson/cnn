#include "hip/hip_runtime.h"
#include "net.h"

#define flat2d(i, j, wid) (i * wid) + j 
#define flat3d(i, j, k, wid, hgt) (hgt * wid * i) + (wid * j) + k

typedef struct GPUClassify {
  size_t *topo;
  double *activs;
  double *wgts;
} GPUClassify_T;

struct Classify {
  size_t size;
  size_t maxNrn;
  GPUClassify_T *dev;
};

struct FLearn {
  size_t stride;
  ImgList_T featrs;
};

size_t CNN_findMax(size_t *, size_t);

FLearn_T *CNN_initFL(size_t numFeat, size_t featWid, size_t featHgt, size_t stride) {
  FLearn_T *net = (FLearn_T *)malloc(sizeof(FLearn_T));
  net->stride = stride;
  net->featrs.numImg = numFeat;
  net->featrs.wid = featWid;
  net->featrs.hgt = featHgt;

  size_t chnlSize = numFeat * featWid * featHgt * sizeof(double);
  hipMalloc((void **)&net->featrs.r, chnlSize);
  hipMalloc((void **)&net->featrs.g, chnlSize);
  hipMalloc((void **)&net->featrs.b, chnlSize);

  return net;
}

__global__ void cuda_initC(GPUClassify_T *net, size_t *topo, size_t netSize) {
  hipMalloc((void **)&net->topo, netSize * sizeof(size_t));
  /* deep copy- host memory can only be freed in host */
  for (int i = 0; i < netSize; i++) {
    net->topo[i] = topo[i];
  }

  size_t totalNrn = 0;
  for (int i = 0; i < netSize; i++) {
    totalNrn += topo[i];
  }
  hipMalloc((void **)&net->activs, totalNrn * sizeof(double));

  size_t totalWgt = 0;
  for (int i = 0; i < netSize - 1; i++) {
    totalWgt += topo[i] * topo[i + 1];
  }
  hipMalloc((void **)&net->wgts, totalWgt * sizeof(double));
}

Classify_T *CNN_initC(size_t *topology, size_t netSize) {
  assert(netSize > 0 && topology != NULL);
  for (int i = 0; i < netSize; i++) {
    assert(topology[i] != 0);
  }

  Classify_T *net = (Classify_T *)malloc(sizeof(Classify_T));
  net->maxNrn = CNN_findMax(topology, netSize);
  net->size = netSize;

  hipMalloc((void **)&net->dev, sizeof(GPUClassify_T));

  /* init topology */
  size_t *topo_d;
  hipMalloc((void **)&topo_d, netSize * sizeof(size_t));
  hipMemcpy(topo_d, topology, netSize * sizeof(size_t), hipMemcpyHostToDevice);

  cuda_initC<<<1,1>>>(net->dev, topo_d, netSize);
  hipDeviceSynchronize();
  hipFree(topo_d);

  return net;
}

__global__ void cuda_freeC(GPUClassify_T *net) {
  hipFree(net->topo);
  hipFree(net->activs);
  hipFree(net->wgts);
}

void CNN_freeC(Classify_T *net) {
  cuda_freeC<<<1,1>>>(net->dev);
  hipDeviceSynchronize();
  hipFree(net->dev);
  free(net);
  hipDeviceReset();
}

__global__ void cuda_testC(GPUClassify_T *net, size_t size) {
	for (size_t i = 0; i < size; i++) {
		printf("  layer %lu:\n", i);
		printf("    numNrn %lu:\n", net->topo[i]);
		for (size_t j = 0; j < net->topo[i]; j++) {
			printf("    neuron %lu (activ: %.2f) weights:\n      ", j, net->activs[flat2d(i, j, size)]);
      if (i != size - 1) {
				for (size_t k = 0; k < net->topo[i + 1]; k++) {
					printf("[%.2f] ", net->wgts[flat3d(i, j, k, size, net->topo[i])]);
				}
      }
			printf("\n");
		}
	}
}

void CNN_testC(Classify_T *net) {
  cuda_testC<<<1,1>>>(net->dev, net->size);
  hipDeviceSynchronize();
}

void CNN_freeFL(FLearn_T *net) {
  hipFree(net->featrs.r);
  hipFree(net->featrs.b);
  hipFree(net->featrs.g);

  free(net);
}

void CNN_freeData(Data_T* input) {
  hipFree(input->lbls);
  hipFree(input->imgDat.r);
  hipFree(input->imgDat.g);
  hipFree(input->imgDat.b);
  hipDeviceReset();
  free(input);
}

size_t CNN_findMax(size_t *arr, size_t len) {
  size_t max = 0;
  for (int i = 0; i < len; i++) {
    if (arr[i] > max) {
      max = arr[i];
    }
  }

  return max;
}
