#include "hip/hip_runtime.h"
#include "net.h"

#define flat2d(i, j, wid) (i * wid) + j
#define flat3d(i, j, k, wid, hgt) (hgt * wid * i) + (wid * j) + k

typedef struct GPUClassify {
  size_t *topo;
  double *activs;
  double *wgts;
} GPUClassify_T;

struct Classify {
  size_t size;
  size_t maxNrn;
  GPUClassify_T *dev;
};

struct FeatLrn {
  NetCfg_T spec;
  double *r;
  double *g;
  double *b;
};

size_t CNN_findMax(size_t *, size_t);

FeatLrn_T *CNN_initFL(NetCfg_T spec) {
  size_t chnlSize = spec.fltrWid * spec.fltrHgt * spec.numFltr * sizeof(double);
  assert(chnlSize != 0);
  FeatLrn_T *net = (FeatLrn_T *)malloc(sizeof(FeatLrn_T));
  net->spec = spec;
  hipMalloc((void **)&net->r, chnlSize);
  hipMalloc((void **)&net->b, chnlSize);
  hipMalloc((void **)&net->g, chnlSize);

  return net;
}

__global__ void cuda_initC(GPUClassify_T *net, size_t *topo, size_t netSize) {
  hipMalloc((void **)&net->topo, netSize * sizeof(size_t));
  /* deep copy- host memory can only be freed in host */
  for (int i = 0; i < netSize; i++) {
    net->topo[i] = topo[i];
  }

  size_t totalNrn = 0;
  for (int i = 0; i < netSize; i++) {
    totalNrn += topo[i];
  }
  hipMalloc((void **)&net->activs, totalNrn * sizeof(double));

  size_t totalWgt = 0;
  for (int i = 0; i < netSize - 1; i++) {
    totalWgt += topo[i] * topo[i + 1];
  }
  hipMalloc((void **)&net->wgts, totalWgt * sizeof(double));
}

Classify_T *CNN_initC(size_t *topology, size_t netSize) {
  assert(netSize > 0 && topology != NULL);
  for (int i = 0; i < netSize; i++) {
    assert(topology[i] != 0);
  }

  Classify_T *net = (Classify_T *)malloc(sizeof(Classify_T));
  net->maxNrn = CNN_findMax(topology, netSize);
  net->size = netSize;

  hipMalloc((void **)&net->dev, sizeof(GPUClassify_T));

  /* init topology */
  size_t *topo_d;
  hipMalloc((void **)&topo_d, netSize * sizeof(size_t));
  hipMemcpy(topo_d, topology, netSize * sizeof(size_t), hipMemcpyHostToDevice);

  cuda_initC<<<1,1>>>(net->dev, topo_d, netSize);
  hipDeviceSynchronize();
  hipFree(topo_d);

  return net;
}

__global__ void cuda_freeC(GPUClassify_T *net) {
  hipFree(net->topo);
  hipFree(net->activs);
  hipFree(net->wgts);
}

void CNN_freeC(Classify_T *net) {
  cuda_freeC<<<1,1>>>(net->dev);
  hipDeviceSynchronize();
  hipFree(net->dev);
  free(net);
  hipDeviceReset();
}

__global__ void cuda_testC(GPUClassify_T *net, size_t size) {
	for (size_t i = 0; i < size; i++) {
		printf("  layer %lu:\n", i);
		printf("    numNrn %lu:\n", net->topo[i]);
		for (size_t j = 0; j < net->topo[i]; j++) {
			printf("    neuron %lu (activ: %.2f) weights:\n      ", j, net->activs[flat2d(i, j, size)]);
      if (i != size - 1) {
				for (size_t k = 0; k < net->topo[i + 1]; k++) {
					printf("[%.2f] ", net->wgts[flat3d(i, j, k, size, net->topo[i])]);
				}
      }
			printf("\n");
		}
	}
}

void CNN_testC(Classify_T *net) {
  cuda_testC<<<1,1>>>(net->dev, net->size);
  hipDeviceSynchronize();
}

void CNN_freeFC(FeatLrn_T *net) {
  hipFree(net->r);
  hipFree(net->g);
  hipFree(net->b);
  free(net);
}

size_t CNN_findMax(size_t *arr, size_t len) {
  size_t max = 0;
  for (int i = 0; i < len; i++) {
    if (arr[i] > max) {
      max = arr[i];
    }
  }

  return max;
}
