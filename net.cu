#include "hip/hip_runtime.h"
#include "net.h"

#define SIG(x) 1.0f/(1.0f + exp(-x))
#define DSIG(x) SIG(x) * (1 - SIG(X))

__global__ void CNN_convolve(Net_T *net, Features_T *kern, double *img) {
  size_t x = FLAT2D(blockIdx.x, threadIdx.x, blockDim.x);
  size_t y = FLAT2D(blockIdx.y, threadIdx.y, blockDim.y);
  size_t z = FLAT2D(blockIdx.z, threadIdx.z, blockDim.z);

  size_t imgWid = net->wid + kern->wid - 1;
  size_t kernPxls = kern->hgt * kern->wid;
  
  if (x < kern->num && y < net->hgt && z < net->wid) {
    double chnlTotal[NUM_CHNL] = { 0.0f, 0.0f, 0.0f };
    for (size_t i = y; i - y < kern->hgt; i++) {
      for (size_t j = z; j - z < kern->wid; j++) {
        for (size_t k = 0; k < NUM_CHNL; k++) {
          chnlTotal[k] += img[FLAT3D(i, j, k, imgWid, NUM_CHNL)] * 
                    kern->imgs[FLAT4D(x, (i - y), (j - z), k, kern->hgt, kern->wid, NUM_CHNL)];
        }
      }
    }

    for (uint8_t i = 0; i < NUM_CHNL; i++) {
      net->imgs[FLAT4D(x, y, z, i, net->hgt, net->wid, NUM_CHNL)] = chnlTotal[i] / kernPxls;
    }
  }
}

__global__ void CNN_normalize(Net_T *net) {
  size_t x = FLAT2D(blockIdx.x, threadIdx.x, blockDim.x);
  if (x < net->num * net->hgt * net->wid * NUM_CHNL && net->imgs[x] < 0) {
    net->imgs[x] = 0;
  }
}

__global__ void CNN_pool(Net_T *net, Pool_T *pool, double *buffer) {
  size_t x = FLAT2D(blockIdx.x, threadIdx.x, blockDim.x);
  size_t y = FLAT2D(blockIdx.y, threadIdx.y, blockDim.y);
  size_t z = FLAT2D(blockIdx.z, threadIdx.z, blockDim.z);

  size_t hgt = net->hgt;
  size_t wid = net->wid;
  size_t poolHgt = ((hgt - pool->winDim) / pool->stride) + 1;
  size_t poolWid = ((wid - pool->winDim) / pool->stride) + 1;

  if (x < net->num && y < hgt && y % pool->stride == 0 && z < wid && z % pool->stride == 0) {
    double chnlMax[NUM_CHNL] = { DBL_MIN, DBL_MIN, DBL_MIN };
    for (size_t i = y; i - y < pool->winDim; i++) {
      for (size_t j = z; j - z < pool->winDim; j++) {
        for (uint8_t k = 0; k < NUM_CHNL; k++) {
          double curPxl = net->imgs[FLAT4D(x, i, j, k, hgt, wid, NUM_CHNL)];
          if(curPxl > chnlMax[k]) {
            chnlMax[k] = curPxl;
          }
        }
      }
    }

    for (uint8_t i = 0; i < NUM_CHNL; i++) {
      buffer[FLAT4D(x, (y / pool->stride), (z / pool->stride), i, poolHgt, poolWid, NUM_CHNL)] = chnlMax[i];
    }
  }

  net->hgt = poolHgt;
  net->wid = poolWid;

  __syncthreads();
  if (x < net->num && y < poolHgt && z < poolWid) {
    for (int i = 0; i < NUM_CHNL; i++) {
      size_t idx = FLAT4D(x, y, z, i, poolHgt, poolWid, NUM_CHNL);
      net->imgs[idx] = buffer[idx];
    }
  }
}


__global__ void CNN_feedForward(Classify_T *cls) {
  for (size_t i = 1; i < cls->numLyr; i++) {
    size_t numNrn = (i == cls->numLyr - 1) ? cls->topo[i] : cls->topo[i] - 1;
    size_t prevLyr = i - 1;
    for (size_t j = 0; j < numNrn; j++) {
      double sum = 0;
      for (size_t k = 0; k < cls->topo[prevLyr]; k++) {
        sum += cls->wgts[FLAT3D(prevLyr, k, j, cls->topo[prevLyr], numNrn)]
             * cls->activs[FLAT2D(prevLyr, k, cls->topo[prevLyr])];
      }
      cls->activs[FLAT2D(i, j, numNrn)] = SIG(sum);
    }
  }
}

__global__ void CNN_testNet(Net_T *net) {
  printf("Num features: %lu\n", net->num);
  printf("Feature wid: %lu\n", net->hgt);
  printf("Feature Hgt: %lu\n", net->wid);

  for (size_t i = 0; i < net->num; i++) {
    printf("Printing netolved feature #%lu:\n", i);
    printf("Red Channel:\n");
    for (size_t j = 0; j < net->hgt; j++) {
      for (size_t k = 0; k < net->wid; k++) {
        printf("%0.2f ", net->imgs[FLAT4D(i, j, k, 0, net->hgt, net->wid, NUM_CHNL)]);
      }
      printf("\n");
    }

    printf("Green Channel:\n");
    for (size_t j = 0; j < net->hgt; j++) {
      for (size_t k = 0; k < net->wid; k++) {
        printf("%0.2f ", net->imgs[FLAT4D(i, j, k, 1, net->hgt, net->wid, NUM_CHNL)]);
      }
      printf("\n");
    }

    printf("Blue Channel:\n");
    for (size_t j = 0; j < net->hgt; j++) {
      for (size_t k = 0; k < net->wid; k++) {
        printf("%0.2f ", net->imgs[FLAT4D(i, j, k, 2, net->hgt, net->wid, NUM_CHNL)]);
      }
      printf("\n");
    }
  }
}

/* TODO: this is bugged- don't call it until you fix */
__global__ void cuda_testClsfier(Classify_T *cls) {
  size_t size = cls->numLyr;
	for (size_t i = 0; i < size; i++) {
		printf("  layer %lu:\n", i);
		printf("    numNrn %lu:\n", cls->topo[i]);
		for (size_t j = 0; j < cls->topo[i]; j++) {
      if (j == cls->topo[i] - 1 && i != 0 && i != size - 1) {
        printf("BIAS ");
      } else {
        printf("     ");
      }
      printf("neuron %lu (activ: %.1f) weights:\n      ", j, cls->activs[FLAT2D(i, j, cls->topo[i])]);
      if (i != size - 1) {
        size_t loopLimit = (i == size - 2) ? cls->topo[i + 1] : cls->topo[i + 1] - 1;
				for (size_t k = 0; k < loopLimit; k++) {
					printf("[%.1f] ", cls->wgts[FLAT3D(i, j, k, cls->topo[i], cls->topo[i + 1])]);
				}
      }
			printf("\n");
		}
	}
}

void CNN_testClsfier(Classify_T *cls) {
  cuda_testClsfier<<<1,1>>>(cls);
  hipDeviceSynchronize();
}

__global__ void cuda_testData(double *imgs, size_t *lbls, size_t idx, size_t hgt, size_t wid) {
  double sum = 0;
  printf("Red Channel:\n");
  for (size_t i = 0; i < hgt; i++) {
    for (size_t j = 0; j < wid; j++) {
      printf("%0.2f ", imgs[FLAT4D(idx, i, j, 0, hgt, wid, NUM_CHNL)]);
      sum += imgs[FLAT4D(idx, i, j, 0, hgt, wid, NUM_CHNL)];
    }
    printf("\n");
  }

  sum = 0;
  printf("Green Channel:\n");
  for (size_t i = 0; i < hgt; i++) {
    for (size_t j = 0; j < wid; j++) {
      printf("%0.2f ", imgs[FLAT4D(idx, i, j, 1, hgt, wid, NUM_CHNL)]);
      sum += imgs[FLAT4D(idx, i, j, 1, hgt, wid, NUM_CHNL)];
    }
    printf("\n");
  }

  sum = 0;
  printf("Blue Channel:\n");
  for (size_t i = 0; i < hgt; i++) {
    for (size_t j = 0; j < wid; j++) {
      printf("%0.2f ", imgs[FLAT4D(idx, i, j, 2, hgt, wid, NUM_CHNL)]);
      sum += imgs[FLAT4D(idx, i, j, 2, hgt, wid, NUM_CHNL)];
    }
    printf("\n");
  }
}

void CNN_testData(Data_T *data, size_t idx) {
  cuda_testData<<<1, 1>>>(data->imgs, data->lbls, idx, data->hgt, data->wid);
  hipDeviceSynchronize();
}
