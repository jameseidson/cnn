#include "hip/hip_runtime.h"
#include "mat.h"

#define SIG(x) (1.0/(1.0 + exp(-x)))
#define DSIG(x) (x * (1 - x))

#define RELU(x) ((x > 0) ? x : 0)
#define DRELU(x) ((double)(x > 0))

#define SE(obs, tru) (0.5 * (tru - obs) * (tru - obs))
#define DSE(obs, tru) (obs - tru)

#define LCG(seed) ((1103515245 * seed + 12345) % INT_MAX)

__global__ void MAT_randomize(double *m, size_t numElm) {
  size_t x = FLAT2D(blockIdx.x, threadIdx.x, blockDim.x);
  if (x < numElm) {
    m[x] = (double)(LCG(x) / (double)INT_MAX / (double)100);
  }
}

__global__ void MAT_setVal(double *m, size_t numElm, double val) {
  size_t x = FLAT2D(blockIdx.x, threadIdx.x, blockDim.x);
  if (x < numElm) {
    m[x] = val;
  }
}

__global__ void MAT_assign(double *src, double *dst, size_t numElm) {
  size_t x = FLAT2D(blockIdx.x, threadIdx.x, blockDim.x);
  if (x < numElm) {
    dst[x] = src[x];
  }
}

__global__ void MAT_ewMul(double *mA, double *mB, double *mC, size_t numElm) {
  size_t x = FLAT2D(blockIdx.x, threadIdx.x, blockDim.x);

  if (x < numElm) {
    mC[x] = mA[x] * mB[x];
  }
}

__global__ void MAT_mvMul(double *mA, double *vB, double *vC, size_t aRows, size_t aCols) {
  size_t r = FLAT2D(blockIdx.x, threadIdx.x, blockDim.x);

  if (r < aRows) {
    double dotProd = 0.0;
    for (size_t i = 0; i < aCols; i++) {
      dotProd += mA[FLAT2D(r, i, aCols)] * vB[i];
    }
    vC[r] = dotProd;
  }
}

__global__ void MAT_ReLU(double *mA, double *mB, size_t numElm) {
  size_t x = FLAT2D(blockIdx.x, threadIdx.x, blockDim.x);

  if (x < numElm) {
    mB[x] = RELU(mA[x]);
  }
}

__global__ void MAT_sigmoid(double *mA, double *mB, size_t numElm) {
  size_t x = FLAT2D(blockIdx.x, threadIdx.x, blockDim.x);

  if (x < numElm) {
    mB[x] = SIG(mA[x]);
  }
}

__global__ void MAT_loss(double *m, size_t numElm, size_t lbl, double *loss) {
  size_t x = FLAT2D(blockIdx.x, threadIdx.x, blockDim.x);

  if (x < numElm) {
    *loss += SE(m[x], (double)(x == lbl));
  }
}

__global__ void MAT_fwdProp(double *mWgt, double *vAct, double *vNxtAct, size_t wRows, size_t wCols, NonLin_T fType) {
  size_t r = FLAT2D(blockIdx.x, threadIdx.x, blockDim.x);

  if (r < wRows) {
    double dotProd = 0.0;
    for (size_t i = 0; i < wCols; i++) {
      dotProd += mWgt[FLAT2D(r, i, wCols)] * vAct[i];
    }
    vNxtAct[r] = (fType == RELU) ? RELU(dotProd) : SIG(dotProd);
  }
}

__global__ void MAT_deltas_out(double *vOut, double *vDelt, size_t numElm, size_t lbl, NonLin_T fType) {
  size_t x = FLAT2D(blockIdx.x, threadIdx.x, blockDim.x);

  if (x < numElm) {
    vDelt[x] = DSE(vOut[x], (double)(x == lbl)) * ((fType == RELU) ? DRELU(vOut[x]) : DSIG(vOut[x]));
  }
}

__global__ void MAT_deltas_hidden(double *vAct, double *vDelt, double *mWgt, double *vNxtDelt, size_t wRows, size_t wCols, NonLin_T fType) {
  size_t c = FLAT2D(blockIdx.x, threadIdx.x, blockDim.x);

  if (c < wCols) {
    double dotProd = 0.0;
    for (size_t i = 0; i < wRows; i++) {
      dotProd += mWgt[FLAT2D(c, i, wRows)] * vNxtDelt[i];
    
    }
    vDelt[c] = dotProd * ((fType == RELU) ? DRELU(vAct[c]) : DSIG(vAct[c]));
  }
}

__global__ void MAT_applyGradient(double *vAct, double *vNxtDelt, double *mWgt, size_t wRows, size_t wCols, double scaleFac) {
  size_t r = FLAT2D(blockIdx.x, threadIdx.x, blockDim.x);
  size_t c = FLAT2D(blockIdx.y, threadIdx.y, blockDim.y);

  if (r < wRows) {
    if (c < wCols) {
      mWgt[FLAT2D(r, c, wCols)] -= scaleFac * vAct[c] * vNxtDelt[r];
    }
  }
}

__global__ void MAT_convolve(double *mA, double *mB, double *mKern, size_t aRows, size_t aCols, size_t kRows, size_t kCols) {
  size_t r = FLAT2D(blockIdx.x, threadIdx.x, blockDim.x);
  size_t c = FLAT2D(blockIdx.y, threadIdx.y, blockDim.y);

  size_t kernElm = kRows * kCols;
  size_t bRows = aRows - kRows + 1;
  size_t bCols = aCols - kCols + 1;

  if (r < bRows && c < bCols) {
    double chnlTotal[NUM_CHNL] = { 0.0, 0.0, 0.0 };
    for (size_t i = r; i - r < kRows; i++) {
      for (size_t j = c; j - c < kCols; j++) {
        for (size_t k = 0; k < NUM_CHNL; k++) {
          chnlTotal[k] += mA[FLAT3D(i, j, k, aRows, NUM_CHNL)] * mKern[FLAT3D((i - r), (j - c), k, kCols, NUM_CHNL)];
        }
      }
    }
    for (size_t i = 0; i < NUM_CHNL; i++) {
      mB[FLAT3D(r, c, i, bCols, NUM_CHNL)] = chnlTotal[i] / kernElm;
    }
  }
}

__global__ void MAT_pool(double *mA, double *mB, size_t aRows, size_t aCols, size_t wDim, size_t stride) {
  size_t r = FLAT2D(blockIdx.x, threadIdx.x, blockDim.x);
  size_t c = FLAT2D(blockIdx.y, threadIdx.y, blockDim.y);

  size_t bCols = ((aCols - wDim) / stride) + 1;
  size_t bRows = ((aRows - wDim) / stride) + 1;

  if (r < bRows && c < bCols) {
    double chnlMax[NUM_CHNL] = { DBL_MIN, DBL_MIN, DBL_MIN };

    for (size_t i = 0; i < wDim; i++) {
      for (size_t j = 0; j < wDim; j++) {
        for (size_t k = 0; k < NUM_CHNL; k++) {
          double curElm = mA[FLAT3D(FLAT2D(r, i, stride), FLAT2D(c, j, stride), k, bCols, NUM_CHNL)];
          if (curElm > chnlMax[k]) {
            chnlMax[k] = curElm;
          }
        }
      }
    }
  }
}

__global__ void MAT_print(double *m, size_t rows, size_t cols, bool is3D) {
  if (is3D) {
    for (uint8_t k = 0; k < NUM_CHNL; k++) {
      printf("Channel %u:\n", k);
      for (size_t i = 0; i < rows; i++) {
        for (size_t j = 0; j < cols; j++) {
          printf("%0.3f ", m[FLAT3D(i, j, k, cols, NUM_CHNL)]);
        }
        printf("\n");
      }
    }
    printf("\n");
  } else if (cols == 1) {
    for (size_t i = 0; i < rows; i++) {
      printf("%0.3f", m[i]);
      printf("\n");
    }
  } else {
    for (size_t i = 0; i < rows; i++) {
      for (size_t j = 0; j < cols; j++) {
        printf("%0.3f ", m[FLAT2D(i, j, cols)]);
      }
      printf("\n");
    }
  }
}
