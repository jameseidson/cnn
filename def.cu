#include "hip/hip_runtime.h"
#include "def.h"

size_t findMax(size_t *, size_t);

__global__ void cuda_initClsfier(Classify_T *cls, size_t *topo, size_t numLyr, size_t maxNrn) {
  cls->numLyr = numLyr;
  cls->maxNrn = maxNrn;

  /* deep copy topo otherwise we must free from host, which is not possible because it's a member of a 
  device-stored struct. yeah, it's pretty gross */
  hipMalloc((void **)&cls->topo, numLyr * sizeof(size_t));
  for (size_t i = 0; i < numLyr; i++) {
    cls->topo[i] = (i == 0 || i == numLyr - 1) ? topo[i] : topo[i] + 1;
  }

  size_t totalNrn = 0;
  for (size_t i = 0; i < numLyr; i++) {
    totalNrn += cls->topo[i];
  }
  hipMalloc((void **)&cls->activs, totalNrn * sizeof(double));
  for (size_t i = 0; i < totalNrn; i++) {
    cls->activs[i] = 0.0f;
  }


  size_t totalWgt = 0;
  for (size_t i = 0; i < numLyr- 1; i++) {
    totalWgt += cls->topo[i] * topo[i + 1];
  }
  hipMalloc((void **)&cls->wgts, totalWgt * sizeof(double));
  for (size_t i = 0; i < totalWgt; i++) {
    cls->wgts[i] = 0.01f;
  }
}

Classify_T *CNN_initClsfier(size_t *topology, size_t numLyr) {
  Classify_T *cls;
  hipMalloc((void **)&cls, sizeof(Classify_T));

  size_t *topo;
  hipMalloc((void **)&topo, numLyr * sizeof(size_t));
  hipMemcpy(topo, topology, numLyr * sizeof(size_t), hipMemcpyHostToDevice);

  cuda_initClsfier<<<1, 1>>>(cls, topo, numLyr, findMax(topology, numLyr));
  hipDeviceSynchronize();

  hipFree(topo);
  return cls;
} __global__ void cuda_initConvlvd(Convlvd_T *conv, Features_T *kern, size_t imgHgt, size_t imgWid,
                                                                    size_t winDim, size_t stride) {
  conv->num = kern->num;

  conv->hgt = imgHgt - kern->hgt + 1;
  conv->wid = imgWid - kern->wid + 1;

  assert(winDim > 0 && winDim < conv->hgt && winDim < conv->wid);
  assert((conv->hgt - winDim) % stride == 0);
  assert((conv->wid - winDim) % stride == 0);

  hipMalloc((void **)&conv->imgs, conv->num * conv->hgt * conv->wid * NUM_CHNL * sizeof(double));

  size_t totalPxls = conv->num * conv->hgt * conv->wid * NUM_CHNL;
  for (size_t i = 0; i < totalPxls; i++) {
    conv->imgs[i] = 0.1f;
  }

  conv->winDim = winDim;
  conv->stride = stride;
}

Convlvd_T *CNN_initConvlvd(Features_T *kern, Data_T *data, size_t winDim, size_t stride) {
  Convlvd_T *conv;
  hipMalloc((void **)&conv, sizeof(Convlvd_T));
  cuda_initConvlvd<<<1, 1>>>(conv, kern, data->hgt, data->wid, winDim, stride);
  hipDeviceSynchronize();

  return conv;
}

__global__ void cuda_initFtrs(Features_T *kern, size_t num, size_t hgt, size_t wid) {
  kern->num = num;
  kern->hgt = hgt;
  kern->wid = wid;

  size_t numImg = num * hgt * wid * NUM_CHNL;
  hipMalloc((void **)&kern->imgs, numImg * sizeof(double));
  for (size_t i = 0; i < numImg; i++) {
    kern->imgs[i] = 0.1f;
  }
}

Features_T *CNN_initFtrs(size_t numFeat, size_t hgt, size_t wid) {
  Features_T *kern;
  hipMalloc((void **)&kern, sizeof(Features_T));
  cuda_initFtrs<<<1, 1>>>(kern, numFeat, hgt, wid);
  hipDeviceSynchronize();

  return kern;
}

__global__ void cuda_freeClsfier(Classify_T *net) {
  hipFree(net->topo);
  hipFree(net->activs);
  hipFree(net->wgts);
}

void CNN_freeClsfier(Classify_T *cls) {
  cuda_freeClsfier<<<1,1>>>(cls);
  hipDeviceSynchronize();
  hipFree(cls);
}

__global__ void cuda_freeConvlvd(Convlvd_T *conv) {
  hipFree(conv->imgs);
}

void CNN_freeConvlvd(Convlvd_T *conv) {
  cuda_freeConvlvd<<<1, 1>>>(conv);
  hipDeviceSynchronize();

  hipFree(conv);
}

__global__ void cuda_freeFtrs(Features_T *kern) {
  hipFree(kern->imgs);
}

void CNN_freeFtrs(Features_T *kern) {
  cuda_freeFtrs<<<1, 1>>>(kern);
  hipDeviceSynchronize();

  hipFree(kern);
}

void CNN_freeData(Data_T *data) {
  hipFree(data->lbls);
  hipFree(data->imgs);
  free(data);
}

size_t findMax(size_t *arr, size_t len) {
  size_t max = 0;
  for (size_t i = 0; i < len; i++) {
    if (arr[i] > max) {
      max = arr[i];
    }
  }

  return max;
}
